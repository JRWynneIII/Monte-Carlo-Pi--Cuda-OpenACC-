#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <time.h>

__global__ void kernel(int* count_d, float* randomnums)
{
	int i;
	double x,y,z;
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	i = tid;
	int xidx = 0, yidx = 0;

	xidx = (i+i);
	yidx = (xidx+1);

	x = randomnums[xidx];
	y = randomnums[yidx];
	z = ((x*x)+(y*y));

	if (z<=1)
		count_d[tid] = 1;
	else
		count_d[tid] = 0;	
}

void CUDAErrorCheck()
{
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{	
		printf("CUDA error : %s (%d)\n", hipGetErrorString(error), error);
		exit(0);
	}
}

int main(int argc,char* argv[])
{
	int niter = 100000;
	float *randomnums;
	double pi;
	hipMalloc((void**)&randomnums, (2*niter)*sizeof(float));
	// Use CuRand to generate an array of random numbers on the device
	int status;
	hiprandGenerator_t gen;
	status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);
	status |= hiprandSetPseudoRandomGeneratorSeed(gen, 4294967296ULL^time(NULL));
	status |= hiprandGenerateUniform(gen, randomnums, (2*niter));
	status |= hiprandDestroyGenerator(gen);
	if (status != HIPRAND_STATUS_SUCCESS)
	{
		printf("CuRand Failure\n");
		exit(EXIT_FAILURE);
	}

	int threads = 1000;
	int blocks = 100;
	int* count_d;
	int *count = (int*)malloc(blocks*threads*sizeof(int));
	unsigned int reducedcount = 0;
	hipMalloc((void**)&count_d, (blocks*threads)*sizeof(int));
	CUDAErrorCheck();
	//one point per thread
	kernel <<<blocks, threads>>> (count_d, randomnums);
	hipDeviceSynchronize();
	CUDAErrorCheck();
	hipMemcpy(count, count_d, blocks*threads*sizeof(int), hipMemcpyDeviceToHost);
	int i = 0;
	//reduce array into int
	for(i = 0; i<niter; i++)
		reducedcount += count[i];
	hipFree(randomnums);
	hipFree(count_d);
	free(count);

	pi = ((double)reducedcount/niter)*4.0;
	printf("Pi: %f\n", pi);

	return 0;
}
