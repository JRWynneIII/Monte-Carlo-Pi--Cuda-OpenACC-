#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <time.h>

__global__ void kernel(int* count_d, float* randomnums)
{
	int i;
	double x,y,z;
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	i = tid;
	int xidx = 0, yidx = 0;

	xidx = (i+i);
	yidx = (xidx+1);

	x = randomnums[xidx];
	y = randomnums[yidx];
	z = ((x*x)+(y*y));

	if (z<=1)
		count_d[tid] = 1;
	else
		count_d[tid] = 0;	
}

__global__ void reduction(int* count_d)
{
	extern __shared__ int sdata[];
	int tid = threadIdx.x;
	__shared__ int blockTotal;
	blockTotal = 0;
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	__syncthreads();
	for (int s = 0; s<16; s++)
		atomicAdd(&sdata[tid], count_d[(i*16)+s]);
		//sdata[tid] += count_d[i+i+s];
	__syncthreads();
	atomicAdd(&blockTotal, sdata[tid]);
	//blockTotal += sdata[tid];
	count_d[blockIdx.x] = blockTotal;
}

void CUDAErrorCheck()
{
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{	
		printf("CUDA error : %s (%d)\n", hipGetErrorString(error), error);
		exit(0);
	}
}

int main(int argc,char* argv[])
{
	int niter = 100000;
	float *randomnums;
	double pi;
	hipMalloc((void**)&randomnums, (2*niter)*sizeof(float));
	// Use CuRand to generate an array of random numbers on the device
	int status;
	hiprandGenerator_t gen;
	status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);
	status |= hiprandSetPseudoRandomGeneratorSeed(gen, 4294967296ULL^time(NULL));
	status |= hiprandGenerateUniform(gen, randomnums, (2*niter));
	status |= hiprandDestroyGenerator(gen);
	if (status != HIPRAND_STATUS_SUCCESS)
	{
		printf("CuRand Failure\n");
		exit(EXIT_FAILURE);
	}

	int threads = 1000;
	int blocks = 100;
	int* count_d;
	int *count = (int*)malloc(blocks*threads*sizeof(int));
	unsigned int reducedcount = 0;

	hipMalloc((void**)&count_d, (blocks*threads)*sizeof(int));
	CUDAErrorCheck();

	kernel <<<blocks, threads>>> (count_d, randomnums);
	hipDeviceSynchronize();
	reduction <<<50, 125, 125*sizeof(int)>>> (count_d);
	hipDeviceSynchronize();
	CUDAErrorCheck();

	hipMemcpy(count, count_d, 100*sizeof(int), hipMemcpyDeviceToHost);
	int i = 0;
	//reduce array into int
	for(i = 0; i<50; i++)
	{
		reducedcount += count[i];
		printf("count[%d]:\t%d\n", i, count[i]);
	}

	hipFree(randomnums);
	hipFree(count_d);
	free(count);

	pi = ((double)reducedcount/niter)*4.0;
	printf("Pi: %f\n", pi);

	return 0;
}
