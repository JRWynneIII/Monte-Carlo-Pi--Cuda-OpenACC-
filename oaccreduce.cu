#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <time.h>

__global__ void kernel(int* count_d, float* randomnums)
{
	int i;
	double x,y,z;
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	i = tid;
	int xidx = 0, yidx = 0;

	xidx = (i+i);
	yidx = (xidx+1);

	x = randomnums[xidx];
	y = randomnums[yidx];
	z = ((x*x)+(y*y));

	if (z<=1)
		count_d[tid] = 1;
	else
		count_d[tid] = 0;	
}

void CUDAErrorCheck()
{
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{	
		printf("CUDA error : %s (%d)\n", hipGetErrorString(error), error);
		exit(0);
	}
}

extern "C" int* launch(int threads, int blocks)
{
	int niter = 230400;
	float *randomnums;
	hipMalloc((void**)&randomnums, (2*niter)*sizeof(float));
	// Use Rand to generate an array of random numbers on the device
	int status;
	hiprandGenerator_t gen;
	status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);
	status |= hiprandSetPseudoRandomGeneratorSeed(gen, 4294967296ULL^time(NULL));
	status |= hiprandGenerateUniform(gen, randomnums, (2*niter));
	status |= hiprandDestroyGenerator(gen);
	if (status != HIPRAND_STATUS_SUCCESS)
	{
		printf("CuRand Failure\n");
		exit(EXIT_FAILURE);
	}

	int* count_d;
	int *count = (int*)malloc(blocks*threads*sizeof(int));

	hipMalloc((void**)&count_d, (blocks*threads)*sizeof(int));
	CUDAErrorCheck();

	kernel <<<blocks, threads>>> (count_d, randomnums);
	hipDeviceSynchronize();
	//reduction <<<blocks, threads, threads*sizeof(int)>>> (count_d);
	hipDeviceSynchronize();
	CUDAErrorCheck();


	hipFree(randomnums);
	free(count);

	return count_d;
}
