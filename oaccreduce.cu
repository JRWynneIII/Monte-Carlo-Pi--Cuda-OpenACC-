#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <time.h>
#include "reduce.h"

__global__ void kernel(int* count_d, float* randomnums)
{
	int i;
	double x,y,z;
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	i = tid;
	int xidx = 0, yidx = 0;

	xidx = (i+i);
	yidx = (xidx+1);

	x = randomnums[xidx];
	y = randomnums[yidx];
	z = ((x*x)+(y*y));

	if (z<=1)
		count_d[tid] = 1;
	else
		count_d[tid] = 0;	
}

__global__ void reduction(int* count_d)
{
	extern __shared__ int sdata[];
	int tid = threadIdx.x;
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	sdata[tid] = count_d[i];
	__syncthreads();
	//reduce all in sdata[] to one int at sdata[0]
	for (int a = blockDim.x/2; a>0 ;a>>=1)
	{
		if (tid<a)
		{
			sdata[tid] += sdata[tid+a];
		}
	__syncthreads();
	}
	if (tid == 0)
	{
		count_d[blockIdx.x] = sdata[0];
	}
}

void CUDAErrorCheck()
{
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{	
		printf("CUDA error : %s (%d)\n", hipGetErrorString(error), error);
		exit(0);
	}
}

extern "C" int* launch(int threads, int blocks)
{
	int niter = 230400;
	float *randomnums;
	hipMalloc((void**)&randomnums, (2*niter)*sizeof(float));
	// Use Rand to generate an array of random numbers on the device
	int status;
	hiprandGenerator_t gen;
	status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);
	status |= hiprandSetPseudoRandomGeneratorSeed(gen, 4294967296ULL^time(NULL));
	status |= hiprandGenerateUniform(gen, randomnums, (2*niter));
	status |= hiprandDestroyGenerator(gen);
	if (status != HIPRAND_STATUS_SUCCESS)
	{
		printf("CuRand Failure\n");
		exit(EXIT_FAILURE);
	}

	int* count_d;
	int *count = (int*)malloc(blocks*threads*sizeof(int));

	hipMalloc((void**)&count_d, (blocks*threads)*sizeof(int));
	CUDAErrorCheck();

	kernel <<<blocks, threads>>> (count_d, randomnums);
	hipDeviceSynchronize();
	//reduction <<<blocks, threads, threads*sizeof(int)>>> (count_d);
	hipDeviceSynchronize();
	CUDAErrorCheck();


	hipFree(randomnums);
	hipFree(count_d);
	free(count);

	return count_d;
}
